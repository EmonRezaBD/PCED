#include "hip/hip_runtime.h"
﻿//Source code for improved Canny
#define _CRT_SECURE_NO_WARNINGS
#define _USE_MATH_DEFINES

//For CUDA
#include "hip/hip_runtime.h"
#include ""

//For CPP
#include <iostream>
#include <stdio.h>
#include <fstream>
#include <vector>
#include <string>
#include <sstream>

//For openCV
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/cudaimgproc.hpp> //for filtering
#include <opencv2/cudafilters.hpp>  //for filtering
#include <opencv2/cudaarithm.hpp> //for abs
#include <opencv2/imgcodecs.hpp>     // Image file reading and writing

//For Thrust
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#define mode 1 // 0 for sobel, 1 for less, 2 for more

void createGaussianKernel(int);
void cannyDector();
void useGaussianBlur();
void getGradientImg();
void nonMaxSuppress();
void lessHysteresisThreshold(int, int);
void moreHysteresisThreshold();
cv::Mat combineImage();

//openCV variables
cv::Mat oriImage, bluredImage, edgeMagImage, edgeAngImage, thinEdgeImage, thresholdImage;
cv::Mat lowTho, highTho, sobelX, sobelY;
int* gaussianMask, maskRad, maskWidth = 0, maskSum = 0;
float sigma = 0.0, avgGradient = 0.0, var = 0.0;

//CUDA variables
cv::cuda::GpuMat gpuImg;

int main()
{	
    //Read image
    cv::Mat combinedImage, tempImg;
    oriImage = cv::imread("F:\\Improved-Canny-master\\image\\lena.jpg", 0);

    if (oriImage.empty())
    {
        printf("Image read failed\n");
        exit(-1);
    }
    gpuImg.upload(oriImage); //uploading in GPU
    std::cout << "Image UpLoading Done!" << std::endl;
    
    int channels = oriImage.channels();
    if (channels == 1) {
        std::cout << "The image is a grayscale image." << std::endl;
    }
    if (channels == 3) {
        std::cout << "The image is a color image (BGR)." << std::endl;
    }




	return 0;
}
